#include "hip/hip_runtime.h"
#include <blixttest/test.hh>
#include <camera/camera.hh>
#include <color_map/color_map_picker.hh>
#include <harmonic_state/harmonic_state.hh>
#include <onebody_density/onebody_density.hh>
#include <ppm_writer/ppm_writer.hh>
#include <ray_integral/ray_integral.hh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/transform.h>
#include <vector/vector.hh>
#include <vector>

const size_t width = 800;
const size_t height = 800;

float min(thrust::device_vector<float> &array) {
        return *thrust::min_element(array.begin(), array.end());
}

float max(thrust::device_vector<float> &array) {
        return *thrust::max_element(array.begin(), array.end());
}

int main() {
        nucray::camera camera(nucray::vector(2.0f, 0.0f, 0.0f),
                              nucray::vector(0.0f, 1.0f, 0.0f),
                              nucray::vector(0.0f, 0.0f, 1.0f),
                              0.5,
                              width,
                              height);
        nucray::onebody_density density(1.0f,
                                        nucray::harmonic_state(0, 1, 0),
                                        nucray::harmonic_state(0, 1, 0));
        nucray::ray_integral integral(100, 0.1, density);
        auto rays = camera.get_rays();
        auto amplitudes = integral.integrate(rays);

        nucray::color_map_picker pick_color(min(amplitudes), max(amplitudes));
        auto color_map = pick_color.get_color_map("blue-red");
        auto pixels = color_map->apply(amplitudes);
        std::vector<nucray::color> host_pixels(pixels.size());
        thrust::copy(pixels.begin(), pixels.end(), host_pixels.begin());
        nucray::ppm_writer ppm_writer;
        ppm_writer.set_pixels(width, height, host_pixels);
        ppm_writer.write("gausian.ppm");
}
